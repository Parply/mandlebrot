#include "hip/hip_runtime.h"
#include <SFML/Config.hpp>
#include <SFML/Graphics.hpp>
#include <SFML/Graphics/PrimitiveType.hpp>
#include <SFML/Graphics/Texture.hpp>
#include <SFML/Graphics/VertexArray.hpp>
#include <SFML/Window/Event.hpp>
#include <SFML/Window/Mouse.hpp>
#include <bits/stdint-uintn.h>
#include <cstddef>
#include <cstdio>
#include <sys/types.h>
#include <math.h>

#include <assert.h>
#include "hip/hip_runtime_api.h"


#define ZOOMRATE 1.2

inline double __host__ __device__ maxd(double a, double b) { return a>b ? a:b; }
inline double __host__ __device__ mind(double a, double b) { return a<b ? a:b; }
const unsigned W=1920,H=1080,Size= W*H*4,max_it=1000;

const unsigned Threads = 512;
const unsigned Blocks  = (W*H + (Threads-1)) / Threads;
extern "C" {

const double cUpperX=1.0,cLowerX=-2.5,cLowerY=-1.0,cUpperY=1.0;
const uint8_t lightsPreInit[360]={
  0,   0,   0,   0,   0,   1,   1,   2, 
  2,   3,   4,   5,   6,   7,   8,   9, 
 11,  12,  13,  15,  17,  18,  20,  22, 
 24,  26,  28,  30,  32,  35,  37,  39, 
 42,  44,  47,  49,  52,  55,  58,  60, 
 63,  66,  69,  72,  75,  78,  81,  85, 
 88,  91,  94,  97, 101, 104, 107, 111, 
114, 117, 121, 124, 127, 131, 134, 137, 
141, 144, 147, 150, 154, 157, 160, 163, 
167, 170, 173, 176, 179, 182, 185, 188, 
191, 194, 197, 200, 202, 205, 208, 210, 
213, 215, 217, 220, 222, 224, 226, 229, 
231, 232, 234, 236, 238, 239, 241, 242, 
244, 245, 246, 248, 249, 250, 251, 251, 
252, 253, 253, 254, 254, 255, 255, 255, 
255, 255, 255, 255, 254, 254, 253, 253, 
252, 251, 251, 250, 249, 248, 246, 245, 
244, 242, 241, 239, 238, 236, 234, 232, 
231, 229, 226, 224, 222, 220, 217, 215, 
213, 210, 208, 205, 202, 200, 197, 194, 
191, 188, 185, 182, 179, 176, 173, 170, 
167, 163, 160, 157, 154, 150, 147, 144, 
141, 137, 134, 131, 127, 124, 121, 117, 
114, 111, 107, 104, 101,  97,  94,  91, 
 88,  85,  81,  78,  75,  72,  69,  66, 
 63,  60,  58,  55,  52,  49,  47,  44, 
 42,  39,  37,  35,  32,  30,  28,  26, 
 24,  22,  20,  18,  17,  15,  13,  12, 
 11,   9,   8,   7,   6,   5,   4,   3, 
  2,   2,   1,   1,   0,   0,   0,   0, 
  0,   0,   0,   0,   0,   0,   0,   0, 
  0,   0,   0,   0,   0,   0,   0,   0, 
  0,   0,   0,   0,   0,   0,   0,   0, 
  0,   0,   0,   0,   0,   0,   0,   0, 
  0,   0,   0,   0,   0,   0,   0,   0, 
  0,   0,   0,   0,   0,   0,   0,   0, 
  0,   0,   0,   0,   0,   0,   0,   0, 
  0,   0,   0,   0,   0,   0,   0,   0, 
  0,   0,   0,   0,   0,   0,   0,   0, 
  0,   0,   0,   0,   0,   0,   0,   0, 
  0,   0,   0,   0,   0,   0,   0,   0, 
  0,   0,   0,   0,   0,   0,   0,   0, 
  0,   0,   0,   0,   0,   0,   0,   0, 
  0,   0,   0,   0,   0,   0,   0,   0, 
  0,   0,   0,   0,   0,   0,   0,   0};

__device__ __constant__ uint8_t lights[sizeof(lightsPreInit)/sizeof(*lightsPreInit)];
}
void __global__ mandlebrot (sf::Uint8 pixels[],
		double UpperX,double LowerX,double UpperY,double LowerY)
{
	unsigned pixno = blockIdx.x * blockDim.x + threadIdx.x;
    	if(pixno >= W*H) return;
	const unsigned Px = pixno % W;
    	const unsigned Py = pixno / W;
	double x0,y0,x,y,xtemp;
	unsigned it,ang;
			x0= ((UpperX-LowerX)*( Px )/W) + LowerX;
			y0=((UpperY-LowerY)*(Py)/H) +LowerY;
			x=0.0;
			y=0.0;
			it=0;
			while (it<max_it && x*x+y*y<=4)
			{
				xtemp = x*x-y*y+x0;
				y = 2*x*y+y0;
				x=xtemp;
				it++;
			}
			ang=(int) 360*sqrt((double)it/(double)max_it);
			pixels[4*(Px+Py*W)] = lights[(ang+120)%360];
			pixels[4*(Px+Py*W)+1] =lights[ang];
			pixels[4*(Px+Py*W)+2] = lights[(ang+240)%360];
;		
			if (it==max_it)
				pixels[4*(Px+Py*W)+3] = 0;
			else
				pixels[4*(Px+Py*W)+3] = 127;



		
	

	
}


void __host__ __device__ zoom(int mousedelta,unsigned mousex,unsigned mousey,
		double & UpperX,double & LowerX,double & UpperY,double & LowerY)
{
	double midX = (UpperX-LowerX)*((double)mousex/(double) W),midY=(UpperY-LowerY)*((double) mousey/(double) H);
	double xInt,yInt,temp;
	if (mousedelta==1)//zoom
	{
		temp = 1.0/(ZOOMRATE*2.0);
	} else if (mousedelta==-1)//unzoom
	{
		temp = ZOOMRATE*2.0;
	}
	xInt = (UpperX -LowerX)*temp;
	yInt = (UpperY-LowerY)*temp;
	UpperX = mind(midX+xInt, cUpperX);
	LowerX = maxd(midX-xInt, cLowerX);
	UpperY = mind(midY+yInt,cUpperY);
	LowerY = maxd(midY-yInt,cLowerY);
}



int main()
{
	double UpperX=1.0,LowerX=-2.5,LowerY=-1.0,UpperY=1.0;
	
	#define PreInit(symbol, from) checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(symbol), &from, sizeof(from)))
    	PreInit(lights, lightsPreInit);
        checkCudaErrors(hipDeviceSetLimit(hipLimitStackSize,2500));
	static sf::Uint8 pixels[Size],*p=NULL;
	checkCudaErrors(hipMalloc((void**)&p, sizeof(pixels))); assert(p!=NULL);
	sf::RenderWindow window(sf::VideoMode(W, H), "Mandlebrot!");
	window.setFramerateLimit(60);
	
    	sf::Texture texture;
	texture.create(W, H); 

	sf::Sprite sprite(texture);

	mandlebrot<<<Blocks,Threads,0>>> (p,UpperX,LowerX,UpperY,LowerY);
	checkCudaErrors(hipMemcpy(pixels, p, sizeof(pixels), hipMemcpyDeviceToHost));
	texture.update(pixels);
	while (window.isOpen())
    {
        	sf::Event event;
        	while (window.pollEvent(event))
        	{
			switch (event.type) {
				case sf::Event::Closed:
					window.close();
					break;
				case sf::Event::MouseWheelScrolled:
					if (event.mouseWheelScroll.wheel == sf::Mouse::VerticalWheel)
					{
						zoom(event.mouseWheelScroll.delta,event.mouseWheelScroll.x,event.mouseWheelScroll.y,UpperX,LowerX,UpperY,LowerY);
						
						mandlebrot<<<Blocks,Threads,0>>> (p,UpperX,LowerX,UpperY,LowerY);
						checkCudaErrors(hipMemcpy(pixels, p, sizeof(pixels), hipMemcpyDeviceToHost));
	texture.update(pixels);
				
					}
        				break;
				default:
					break;

			}
        	}

        	window.clear();
        	window.draw(sprite);
        	window.display();
    	}
	checkCudaErrors(hipFree(p));


    	return 0;
}
